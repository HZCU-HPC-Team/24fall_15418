#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float *A, float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) {
        C[i] = A[i] + B[i];
    }
}

int main(){
    int N = 10;
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A;
    float *d_B;
    float *d_C;
    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}